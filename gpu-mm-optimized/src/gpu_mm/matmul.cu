#include "hip/hip_runtime.h"
#include <torch/extension.h>

__global__ void matmul_kernel(const float* A, const float* B, float* C, int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < N) {
        float sum = 0.0f;
        for (int i = 0; i < K; ++i)
            sum += A[row * K + i] * B[i * N + col];
        C[row * N + col] = sum;
    }
}

void matmul_launcher(torch::Tensor A, torch::Tensor B, torch::Tensor C) {
    const int M = A.size(0);
    const int K = A.size(1);
    const int N = B.size(1);

    const dim3 threads(16, 16);
    const dim3 blocks((N + 15) / 16, (M + 15) / 16);

    matmul_kernel<<<blocks, threads>>>(
        A.data_ptr<float>(), B.data_ptr<float>(), C.data_ptr<float>(),
        M, N, K
    );
}
